#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include <opencv2/opencv.hpp>
#include <time.h>

//variable for cuda
unsigned char *d_orgImage,*d_bicImage;
int *d_orgWidthStep,*d_bicWidthStep;
int *d_width_t,*d_height_t,*d_width,*d_height;
int *d_ratiox,*d_ratioy;

__device__ float caculateMatrix(float *su,float *sv , unsigned char B[][4]){
	float BC[4]={0};
	float ABC=0;
	for(int i=0;i<4;i++){
		for(int j=0;j<4;j++){
			BC[i] += ((float)B[i][j])*su[j];			
		}
	}
	for(int i=0;i<4;i++){
		ABC+= sv[i]*BC[i];
	}
	return ABC;
}

__global__ void scaleInterpolate(unsigned char *image, unsigned char *image_bic, int *pwidthStep, int *pwidthStep_bic, int *pwidth_t, int *pheight_t, int *pwidth, int *pheight, int *pratiox, int *pratioy){
	int part = blockDim.x * blockIdx.x + threadIdx.x;
	int widthStep=*pwidthStep, widthStep_bic=*pwidthStep_bic;
	int width_t=*pwidth_t, height_t=*pheight_t;
	int width=*pwidth, height=*pheight;
	int ratiox=*pratiox, ratioy=*pratioy;
	int i,j,k,l;
	int bicx,bicy;
	float px,py;
	float deltax[4],deltay[4];
	float su[4],sv[4];
	unsigned char BMat[4][4]={0};
	unsigned char GMat[4][4]={0};
	unsigned char RMat[4][4]={0};
	float Bv,Gv,Rv;
	if(part < width_t*height_t)
	{
	i = part%width_t;
	j = part/width_t;
	px = (float)i;
	py = (float)j;
//scaling
	px=px/(float)ratiox;
	py=py/(float)ratioy;
	if(px > width-1 )px=(float)width-1;
	if(py > height-1)py=(float)height-1;			
//interpolation
	if((int)floor(px)<width && (int)floor(py)< height && (int)floor(px)>=0 && (int)floor(py)>=0)
	{
//bicubic
		bicx = (int)floor(px);
		bicy = (int)floor(py);
		if((int)floor(px)==0)bicx=1;
		if((int)floor(py)==0)bicy=1;
		if((int)floor(px)>=width-3) bicx=width-3;
		if((int)floor(py)>=height-3) bicy=height-3;
		deltax[0] = fabs(px- (bicx-1) );
		deltax[1] = fabs(px-bicx);
		deltax[2] = fabs(px-(bicx+1));
		deltax[3] = fabs(px-(bicx+2));
		deltay[0] = fabs(py-(bicy-1));
		deltay[1] = fabs(py-bicy);
		deltay[2] = fabs(py-(bicy+1));
		deltay[3] = fabs(py-(bicy+2));
		for(k=0;k<4;k++){
			if(  (int)floor(deltax[k])==0  ){
				su[k]=1-2*pow(deltax[k],2)+pow(deltax[k],3);
			}
			else if(  (int)floor(deltax[k])==1  ){
				su[k]=4- 8*deltax[k]+ 5*pow(deltax[k],2) - pow(deltax[k],3);
			}
			else su[k]=0;
			if(  (int)floor( deltay[k])==0  ){
				sv[k]=1-2*pow(deltay[k],2)+pow(deltay[k],3);
			}
			else if(  (int)floor( deltay[k])==1  ){
				sv[k]=4- 8*deltay[k]+ 5*pow(deltay[k],2) - pow(deltay[k],3);
			}
			else sv[k]=0;
		}
		for(k=0;k<4;k++){
			for(l=0;l<4;l++){
				BMat[k][l] = image[ (bicy+k-1) *widthStep + (bicx+l-1)*3   ];
				GMat[k][l] = image[ (bicy+k-1) *widthStep + (bicx+l-1)*3 +1];
				RMat[k][l] = image[ (bicy+k-1) *widthStep + (bicx+l-1)*3 +2];
			}
		}
		Bv = caculateMatrix(su,sv,BMat);
		Gv = caculateMatrix(su,sv,GMat);
		Rv = caculateMatrix(su,sv,RMat);
		if(Bv > 255)Bv=255;
		else if(Bv<0)Bv=0;
		if(Gv > 255)Gv=255;
		else if(Gv<0)Gv=0;
		if(Rv > 255)Rv=255;
		else if(Rv<0)Rv=0;
		image_bic[j*widthStep_bic + i*3   ]=(int)Bv;
		image_bic[j*widthStep_bic + i*3 +1]=(int)Gv;
		image_bic[j*widthStep_bic + i*3 +2]=(int)Rv;
	}
	}
}

void transform(IplImage *image){
	IplImage *image_bic;
	int ratiox = 2,ratioy = 2;
	int width_t = image->width*ratiox;
	int height_t = image->height*ratioy;
	CvSize ImageSize1 = cvSize(width_t , height_t);
	image_bic = cvCreateImage(ImageSize1,IPL_DEPTH_8U,3);
	
	//allocate cuda memory
	hipMalloc((void**)&d_orgImage, image->imageSize);
	hipMalloc((void**)&d_bicImage, image_bic->imageSize);
	hipMalloc((void**)&d_orgWidthStep, sizeof(int));
	hipMalloc((void**)&d_bicWidthStep, sizeof(int));
	hipMalloc((void**)&d_width_t, sizeof(int));
	hipMalloc((void**)&d_height_t, sizeof(int));
	hipMalloc((void**)&d_width, sizeof(int));
	hipMalloc((void**)&d_height, sizeof(int));
	hipMalloc((void**)&d_ratiox, sizeof(int));
	hipMalloc((void**)&d_ratioy, sizeof(int));
	//copy memory from cpu to cuda
	hipMemcpy(d_orgImage, image->imageData, image->imageSize, hipMemcpyHostToDevice);
	hipMemcpy(d_orgWidthStep, &image->widthStep, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_bicWidthStep, &image_bic->widthStep, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_width_t, &width_t, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_height_t, &height_t, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_width, &image->width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_height, &image->height, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ratiox, &ratiox, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ratioy, &ratioy, sizeof(int), hipMemcpyHostToDevice);
	// Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (width_t*height_t + threadsPerBlock - 1) / threadsPerBlock;
    scaleInterpolate<<<blocksPerGrid, threadsPerBlock>>>(d_orgImage, d_bicImage, d_orgWidthStep, d_bicWidthStep, d_width_t, d_height_t, d_width, d_height, d_ratiox, d_ratioy);
	//copy result from cuda to cpu
	hipMemcpy(image_bic->imageData, d_bicImage, image_bic->imageSize, hipMemcpyDeviceToHost);

	cvShowImage("bicubic",image_bic);
	cvSaveImage("bicubic.tif",image_bic);
}

int main(void){	
	clock_t clockbegin,clockend;
	clockbegin = clock();
	IplImage *image=cvLoadImage("test4.tif");
	transform(image);
	clockend = clock();
	printf("time spend: %ld ms\n",clockend - clockbegin);
	//cvShowImage("QQ",image);
	//cvSaveImage("QQ.tif",image);
	cvWaitKey(0);
	return 0;
}
